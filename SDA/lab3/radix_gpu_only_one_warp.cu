#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/scan.h>

#define N 33
#define BLOCK_SIZE 64

int get_nth_digit(int num, int digit) {
    int abs_num = abs(num);
    int divisor = 1;

    for (int i = 0; i < digit; i++) {
        divisor *= 10;
    }

    return (abs_num / divisor) % 10;
}

void count_sort(int* array, int digit, size_t n) {
    int count[10] = {0};
    int* tempMap = (int*)malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) {
        int nth_digit = get_nth_digit(array[i], digit);
        count[nth_digit]++;
    }

    // compute the offsets inside the count arr
    for (int i = 1; i < 10; i++) {
        count[i] += count[i - 1];
    }

    for (int i = n-1; i >= 0; i--) {
        int nth_digit = get_nth_digit(array[i], digit);

        tempMap[count[nth_digit] - 1] = array[i];
        count[nth_digit] -= 1;
    }

    for (int i = 0; i < n; i++) {
        array[i] = tempMap[i];
    }

    free(tempMap);
}

void radix(int* array, size_t n) {
    int maxNumber = 0;
    for (int i = 0; i < n; i++) maxNumber = fmax(maxNumber, array[i]);
    int maxDigits = ceil(log10(maxNumber));

    for (int i = 0; i < maxDigits; i++) {
        count_sort(array, i, n);
    }
}

void init_array(int* arr, int n) {
    for (int i = 0; i < n; i++) {
        arr[i] = rand() % 1000;
    }
}

__global__ void count_kernel(int* arr, int* histogram, int n, int divisor) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        int abs_num = abs(arr[i]);
        int nth_digit = (abs_num / divisor) % 10;

        atomicAdd(&histogram[nth_digit], 1);
    }
}

__global__ void count_sort_kernel(int* arr, int* offsets, int n, int divisor, int* temp) {
    int i = (n - 1) - (blockIdx.x * blockDim.x + threadIdx.x);

    if (i >= 0) {
        int abs_num = abs(arr[i]);
        int nth_digit = (abs_num / divisor) % 10;

        int pos = atomicSub(&offsets[nth_digit], 1) - 1;

        temp[pos] = arr[i];
    }
}

__global__ void max_num_kernel(int* arr, int n, int* max) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        atomicMax(max, arr[i]);
    }
}

void check_order(int* arr, int n) {
    for (int i = 0; i < (n - 1); i++) {
        if(arr[i] > arr[i+1]) {
            printf("Failed at: %d > %d", arr[i], arr[i+1]);
            printf("\n\nTEST FAILED\n\n");
            return;
        }
    }

    printf("\n\nTEST PASSED\n\n");
}

int main() {
    int* h_arr = (int*)malloc(N * sizeof(int));
    int* h_temp = (int*)malloc(N * sizeof(int));
    int* h_histogram = (int*)malloc(10 * sizeof(int));
    int* h_max_num = (int*)calloc(1, sizeof(int));

    init_array(h_arr, N);

    int* d_arr;
    int* d_histogram;
    int* d_temp;
    int* d_max_num;

    hipMalloc(&d_arr, N * sizeof(int));
    hipMalloc(&d_temp, N * sizeof(int));
    hipMalloc(&d_histogram, 10 * sizeof(int));
    hipMalloc(&d_max_num, 1 * sizeof(int));

    hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_histogram, h_histogram, 10 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_max_num, h_max_num, 1 * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    max_num_kernel<<<gridDim, blockDim>>>(d_arr, N, d_max_num);
    hipMemcpy(h_max_num, d_max_num, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_max_num);

    int max_digits = 1;

    while (*h_max_num >= 10) {
        *h_max_num /= 10;
        max_digits++;
    }

    for (int digit = 0; digit < max_digits; digit++) {
        int divisor = pow(10, digit);

        hipMemset(d_histogram, 0, 10 * sizeof(int));
        count_kernel<<<gridDim, blockDim>>>(d_arr, d_histogram, N, divisor);
        hipDeviceSynchronize();

        hipMemcpy(h_histogram, d_histogram, 10 * sizeof(int), hipMemcpyDeviceToHost);
        for (int i = 1; i < 10; i++) {
            h_histogram[i] += h_histogram[i - 1];
        }

        hipMemcpy(d_histogram, h_histogram, 10 * sizeof(int), hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        count_sort_kernel<<<gridDim, blockDim>>>(d_arr, d_histogram, N, divisor, d_temp);
        hipDeviceSynchronize();

        hipMemcpy(d_arr, d_temp, N * sizeof(int), hipMemcpyDeviceToDevice);
    }

    hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);
    check_order(h_arr, N);

    printf("Sorted array: \n");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    free(h_max_num);
    free(h_histogram);
    free(h_arr);
    free(h_temp);

    hipFree(d_max_num);
    hipFree(d_histogram);
    hipFree(d_arr);
    hipFree(d_temp);

    return 0;
}
