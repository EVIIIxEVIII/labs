#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 32
#define N 4

void __global__ compute_offset_table(int* arr, int n, int* global, int chunk) {
    __shared__ int local_offset_map[4];
    __shared__ int sum;

    int gId = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    if (tid == 0) sum = 0;
    if (tid < 4) local_offset_map[tid] = 0;
    __syncthreads();

    if (gId < n) {
        int digit = (arr[gId] >> (2 * chunk)) & 3;
        atomicAdd(&local_offset_map[digit], 1);
    }
    __syncthreads();

    if (tid < 4) {
        atomicAdd(&global[tid], local_offset_map[tid]);
    }
    __syncthreads();

    if (tid < 4) {
        atomicAdd(&sum, local_offset_map[tid]);
    }
    __syncthreads();
}

void init_array(int* arr, int n) {
    for (int i = 0; i < n; i++) {
        arr[i] = rand() % 1000;
    }
}

int main() {
    int* h_arr = (int*)malloc(N * sizeof(int));
    int* h_res = (int*)malloc(4 * sizeof(int));

    init_array(h_arr, N);

    printf("Initial array: \n");
    for (int i = 0; i < N; i++) {
       printf("%b ", h_arr[i]);
    }
    printf("\n");

    printf("First chunk digits: \n");
    for (int i = 0; i < N; i++) {
        printf("%d ", (h_arr[i] >> (2 * 0)) & 3);
    }
    printf("\n");

    int* d_arr;
    int* d_res;

    hipMalloc(&d_arr, N * sizeof(int));
    hipMalloc(&d_res, N * sizeof(int));

    hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    compute_offset_table<<<gridDim, blockDim>>>(d_arr, N, d_res, 0);

    hipMemcpy(h_res, d_res, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Offset map: \n");
    for (int i = 0; i < 4; i++) {
       printf("%d ", h_res[i]);
    }
    printf("\n");


}
